#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "helper.h"
extern "C"{
#include "hostFE.h"
}

__global__ void convKernel(float *outputImage, float *inputImage, float *filter, int imageWidth, int imageHeight, int filterWidth){
    
    int halfFilterWidth = filterWidth / 2;
    float sum = 0;
    int x = threadIdx.x;
    int y = blockIdx.x;

	int imageLTIdx = (y - halfFilterWidth) * imageWidth + x - halfFilterWidth;
	int filterLTIdx = 0;
    for (int r = -halfFilterWidth; r <= halfFilterWidth; r++)
	{
		if (y + r >= 0 && y + r < imageHeight)
		{
			int imageIdx = imageLTIdx;
			int filterIdx = filterLTIdx;
			for (int c = -halfFilterWidth; c <= halfFilterWidth; c++, imageIdx++, filterIdx++)
			{
				if (x + c >= 0 && x + c < imageWidth)
				{
					float factor = filter[filterIdx];
					if (factor != 0.0)
					{
						sum += (inputImage[imageIdx] * factor);
					}
				}
			}
		}
		imageLTIdx += imageWidth;
		filterLTIdx += filterWidth;
    }
    outputImage[y * imageWidth + x] = sum;
}

extern "C"
void hostFE(int filterWidth, float *filter, int imageHeight, int imageWidth,
            float *inputImage, float *outputImage, cl_device_id *device,
            cl_context *context, cl_program *program)
{
    int filterSize = filterWidth * filterWidth * sizeof(float);
    int imageSize = imageHeight * imageWidth * sizeof(float);

    float *d_filter;
    float *d_inputImage;
    float *d_outputImage;
	
	//Allocate
    hipMalloc(&d_filter, filterSize);
    hipMalloc(&d_inputImage, imageSize);
    hipMalloc(&d_outputImage, imageSize);

	//Copy H->D
    hipMemcpy(d_filter, filter, filterSize, hipMemcpyHostToDevice);
    hipMemcpy(d_inputImage, inputImage, imageSize, hipMemcpyHostToDevice);

    int numThreads = imageWidth;
    int numBlocks = imageHeight;
    convKernel<<<numBlocks, numThreads>>>(d_outputImage, d_inputImage, d_filter, imageWidth, imageHeight, filterWidth);

	//Copy D->H
    hipMemcpy(outputImage, d_outputImage, imageSize, hipMemcpyDeviceToHost);

    hipFree(d_filter);
    hipFree(d_inputImage);
    hipFree(d_outputImage);
}